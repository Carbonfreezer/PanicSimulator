#include "hip/hip_runtime.h"
#include "ContinuityEquationSolver.h"
#include <cassert>
#include "CudaHelper.h"
#include <>
#include <math.h>
#include "TransferHelper.h"
#include "DataBase.h"




void ContinuityEquationSolver::PrepareSolver()
{
	m_gradientIconal.PreprareModule();
	m_resultBuffer = TransferHelper::ReserveFloatMemory();
}


__global__ void IntegrateCuda(float timePassed,  size_t strides, float* density,  
								float* derivativeX, float* derivativeY,
							float* result)
{
	__shared__ float xDiv[gBlockSize + 2][gBlockSize + 2];
	__shared__ float yDiv[gBlockSize + 2][gBlockSize + 2];

	

	// Prefill the data.

	// We keep tack of the pixel  we are responsible for.
	int xOrigin = threadIdx.x + gBlockSize * blockIdx.x + 1;
	int yOrigin = threadIdx.y + gBlockSize * blockIdx.y + 1;

	int xScan = threadIdx.x + 1;
	int yScan = threadIdx.y + 1;

	float factor;
	
	float xDivPure;
	float yDivPure;
	float localDensity;

	float xGrad, yGrad;

	xGrad = derivativeX[xOrigin + yOrigin * strides];
	yGrad = derivativeY[xOrigin + yOrigin * strides];


	// If both are zero we get an error here.
	factor = 1.0f / (xGrad * xGrad + yGrad * yGrad + FLT_EPSILON);
	localDensity = density[xOrigin + yOrigin * strides];


	// Deal with the inf * zero situation.
	if (factor == 0.0f)
	{
		xDivPure = 0.0f;
		yDivPure = 0.0f;
	}
	else
	{
		xDivPure = xGrad * factor;
		yDivPure = yGrad * factor;
	}
	
	xDiv[xScan][yScan] = xDivPure * localDensity;
	yDiv[xScan][yScan] = yDivPure * localDensity;

	// Copy over the border lines, we do not need the corner elements.
	// We also do not need the density on the border lines.
	if (threadIdx.x == 0)
	{
		xGrad = derivativeX[(xOrigin - 1) + yOrigin * strides];
		yGrad = derivativeY[(xOrigin - 1) + yOrigin * strides];
		
		factor  = density[(xOrigin - 1) + yOrigin * strides] /(xGrad * xGrad + yGrad * yGrad + FLT_EPSILON);

		if (factor == 0.0f)
		{
			xDiv[xScan - 1][yScan] = 0.0f;
			yDiv[xScan - 1][yScan] = 0.0f;
		}
		else
		{
			xDiv[xScan - 1][yScan] = xGrad * factor;
			yDiv[xScan - 1][yScan] = yGrad * factor;
		}
		
	}
		
	if (threadIdx.x == 31)
	{
		xGrad = derivativeX[(xOrigin + 1) + yOrigin * strides];
		yGrad = derivativeY[(xOrigin + 1) + yOrigin * strides];
		
		factor = density[(xOrigin + 1) + yOrigin * strides] / (xGrad * xGrad + yGrad * yGrad  + FLT_EPSILON);

		if (factor == 0.0f)
		{
			xDiv[xScan + 1][yScan] = 0.0f;
			yDiv[xScan + 1][yScan] = 0.0f;

		}
		else
		{
			xDiv[xScan + 1][yScan] = xGrad * factor;
			yDiv[xScan + 1][yScan] = yGrad * factor;

		}
	}
		
	if (threadIdx.y == 0)
	{
		xGrad = derivativeX[(xOrigin)+(yOrigin - 1) * strides];
		yGrad = derivativeY[(xOrigin)+(yOrigin - 1) * strides];

		factor = density[(xOrigin)+(yOrigin - 1) * strides] / (xGrad * xGrad + yGrad * yGrad + FLT_EPSILON);

		if (factor == 0.0f)
		{
			xDiv[xScan][yScan - 1] = 0.0f;
			yDiv[xScan][yScan - 1] = 0.0f;

		}
		else
		{
			xDiv[xScan][yScan - 1] = xGrad * factor;
			yDiv[xScan][yScan - 1] = yGrad * factor;

		}
	}
		
	if (threadIdx.y == 31)
	{
		xGrad = derivativeX[(xOrigin)+(yOrigin + 1) * strides];
		yGrad = derivativeY[(xOrigin)+(yOrigin + 1) * strides];

		factor = density[(xOrigin)+(yOrigin + 1) * strides] / (xGrad * xGrad + yGrad * yGrad + FLT_EPSILON);

		if (factor == 0.0f)
		{
			xDiv[xScan][yScan + 1] = 0.0f;
			yDiv[xScan][yScan + 1] = 0.0f;
		}
		else
		{
			xDiv[xScan][yScan + 1] = xGrad * factor;
			yDiv[xScan][yScan + 1] = yGrad * factor;
		}
	
	}
		

	__syncthreads();

	
	float xDerivative;
	if (xDivPure >= 0.0f)
		xDerivative = (xDiv[xScan + 1][yScan] - xDiv[xScan][yScan]) / (gCellSize);
	else
		xDerivative = (xDiv[xScan][yScan] - xDiv[xScan - 1][yScan]) / (gCellSize);

	
	float yDerivative;
	if (yDivPure >= 0.0f)
		yDerivative = (yDiv[xScan][yScan + 1] - yDiv[xScan][yScan]) / (gCellSize);
	else
		yDerivative = (yDiv[xScan][yScan] - yDiv[xScan ][yScan - 1]) / (gCellSize);


	float finalValue  = localDensity + timePassed * (xDerivative + yDerivative);
	finalValue = fmaxf(0.0f, finalValue);
	finalValue = fminf(gMaximumDensity, finalValue);

	result[xOrigin + yOrigin * strides] = finalValue;
	
	
	
}


void ContinuityEquationSolver::IntegrateEquation(float timePassed, FloatArray density,  FloatArray timeToDestination, DataBase* dataBase)
{

	UnsignedArray wallData = dataBase->GetWallData();
	// First we need the gradient of the iconal equation.
	m_gradientIconal.ComputeGradient(timeToDestination, wallData);

	FloatArray gradX = m_gradientIconal.GetXComponent();
	FloatArray gradY = m_gradientIconal.GetYComponent();
	assert(gradX.m_stride == gradY.m_stride);
	assert(gradX.m_stride == density.m_stride);
	assert(gradX.m_stride == m_resultBuffer.m_stride);
	assert(gradX.m_stride == wallData.m_stride);

	
	IntegrateCuda CUDA_DECORATOR_LOGIC (timePassed, gradX.m_stride, density.m_array,  gradX.m_array, gradY.m_array, m_resultBuffer.m_array);

	// TODO: Make double buffer later on.

	TransferHelper::CopyDataFromTo(m_resultBuffer, density);
		
}

