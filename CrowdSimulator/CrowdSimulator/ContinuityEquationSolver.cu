#include "hip/hip_runtime.h"
#include "ContinuityEquationSolver.h"
#include <cassert>
#include "CudaHelper.h"
#include <>
#include <math.h>
#include "TransferHelper.h"
#include "DataBase.h"

void ContinuityEquationSolver::PrepareSolver()
{
	m_gradientIconal.PreprareModule();
	m_specialXDerivative.PreprareModule();
	m_specialYDerivative.PreprareModule();

	m_premultipliedGradientX = TransferHelper::ReserveFloatMemory();
	m_premultipliedGradientY = TransferHelper::ReserveFloatMemory();

	m_blockedElements = TransferHelper::ReserveUnsignedMemory();
}

__global__ void Multiply(float* gradientXSource, float* gradientYSource, float* gradientXDestination, float* gradientYDestination, 
		size_t gradientStride, float* densityArray, size_t densityStride, 
	float* velocityArray, size_t velocityStride)
{
	// We keep tack of the pixel  we are responsible for.
	int xOrigin = threadIdx.x + gBlockSize * blockIdx.x + 1;
	int yOrigin = threadIdx.y + gBlockSize * blockIdx.y + 1;

	float density = densityArray[xOrigin + yOrigin * densityStride];
	float velocity = velocityArray[xOrigin + yOrigin * velocityStride];
	float factor = density * velocity * velocity;

	gradientXDestination[xOrigin + yOrigin * gradientStride] = gradientXSource[xOrigin + yOrigin * gradientStride] * factor;
	gradientYDestination[xOrigin + yOrigin * gradientStride] = gradientYSource[xOrigin + yOrigin * gradientStride] * factor;
}

__global__ void IntegrateEuler(float timePassed, float* density, size_t densityStride, float* xComponent, float* yComponent, size_t componentStride)
{
	// We keep tack of the pixel  we are responsible for.
	int xOrigin = threadIdx.x + gBlockSize * blockIdx.x + 1;
	int yOrigin = threadIdx.y + gBlockSize * blockIdx.y + 1;


	float sum = xComponent[xOrigin + yOrigin * componentStride] + yComponent[xOrigin + yOrigin * componentStride];
	
	float accumulator = density[xOrigin + yOrigin * densityStride];
	accumulator += timePassed * sum;
	accumulator = fmaxf(accumulator, 0.0f);
	accumulator = fminf(accumulator, gMaximumDensity);

	density[xOrigin + yOrigin * densityStride] = accumulator;
	
}

__global__ void LogicalOr(unsigned int* dataA, size_t strideA, unsigned int* dataB, size_t strideB, unsigned int* destination, size_t destinationStride)
{
	int xOrigin = threadIdx.x + gBlockSize * blockIdx.x + 1;
	int yOrigin = threadIdx.y + gBlockSize * blockIdx.y + 1;

	destination[xOrigin + yOrigin * destinationStride] = dataA[xOrigin + yOrigin * strideA] || dataB[xOrigin + yOrigin * strideB];
}

void ContinuityEquationSolver::IntegrateEquation(float timePassed, FloatArray density, FloatArray velocity,
	FloatArray timeToDestination, DataBase* dataBase)
{

	// First we need the gradient of the iconal equation.
	m_gradientIconal.ComputeGradient(timeToDestination, dataBase->GetWallData());

	// We need a logical or for the wall and the despawn data for the divergence computation later on.
	// We do this in every update, because we do not want to rely on the fact that the data in the base remains constant.
	LogicalOr CUDA_DECORATOR_LOGIC (dataBase->GetWallData().m_array, dataBase->GetWallData().m_stride, dataBase->GetDespawnData().m_array, dataBase->GetDespawnData().m_stride,
		m_blockedElements.m_array, m_blockedElements.m_stride);
	
	FloatArray gradX = m_gradientIconal.GetXComponent();
	FloatArray gradY = m_gradientIconal.GetYComponent();

	assert(gradX.m_stride == gradY.m_stride);
	assert(gradX.m_stride == m_premultipliedGradientX.m_stride);
	assert(m_premultipliedGradientY.m_stride == m_premultipliedGradientX.m_stride);
	// Hard limit iterations.
	if (timePassed > 10.0f * gMaximumStepsizeContinuitySolver)
		timePassed = 10.0f * gMaximumStepsizeContinuitySolver;
	
	bool endOfIterationReached = false;
	do
	{
		float timeStep;

		if (timePassed <= gMaximumStepsizeContinuitySolver)
		{
			timeStep = timePassed;
			endOfIterationReached = true;
		}
		else
		{
			timeStep = gMaximumStepsizeContinuitySolver;
			timePassed -= timeStep;
		}

		// Pre multiply the gradient with the density and velocity fields.
		Multiply CUDA_DECORATOR_LOGIC(gradX.m_array, gradY.m_array, m_premultipliedGradientX.m_array, m_premultipliedGradientY.m_array,
			gradX.m_stride, density.m_array, density.m_stride, velocity.m_array, velocity.m_stride);


		// Now we can compute the gradients of both fields needed for the final integration step.
		m_specialXDerivative.ComputeGradientXForDivergence(m_premultipliedGradientX, m_blockedElements);
		m_specialYDerivative.ComputeGradientYForDivergence(m_premultipliedGradientY, m_blockedElements);

		// We need the x component of the x derivative and the y component of the y derivative.
		FloatArray xComponent = m_specialXDerivative.GetXComponent();
		FloatArray yComponent = m_specialYDerivative.GetYComponent();

		// Now we can integrate the equation of motion.
		assert(xComponent.m_stride == yComponent.m_stride);

		
		IntegrateEuler CUDA_DECORATOR_LOGIC(timeStep, density.m_array, density.m_stride, xComponent.m_array, yComponent.m_array, xComponent.m_stride);

		
	} while (!endOfIterationReached);

	

	
}

