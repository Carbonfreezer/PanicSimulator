#include "hip/hip_runtime.h"
#include "DensityManager.h"
#include "GlobalConstants.h"
#include "CudaHelper.h"
#include <cassert>
#include <>
#include <math.h>
#include "DataBase.h"
#include "VisualizationHelper.h"


void DensityManager::InitializeManager(DataBase* dataBase)
{
	m_density = TransferHelper::ReserveFloatMemory();
	TransferHelper::CopyDataFromTo(dataBase->GetInitialDensityData(), m_density);
	m_continuitySolver.PrepareSolver();
}

__global__ void ApplyConditions(float* densityBuffer, size_t strideDensity,  float* spawnArea, size_t strideSpawn,
                    unsigned int* despawnData, size_t despawnStride)
{
	int xRead = threadIdx.x + blockIdx.x * blockDim.x + 1;
	int yRead = threadIdx.y + blockIdx.y * blockDim.y + 1;


	// Deal with the border cases.
	if (xRead == 1)
		densityBuffer[yRead * strideDensity] = 0.0f;
	if (yRead == 1)
		densityBuffer[xRead] = 0.0f;
	if (xRead == gGridSizeExternal - 2)
		densityBuffer[(gGridSizeExternal - 1) + yRead * strideDensity] = 0.0f;
	if (yRead == gGridSizeExternal - 2)
		densityBuffer[xRead + (gGridSizeExternal - 1) * strideDensity] = 0.0f;


	// Special fix for the corner cases.
	if ((xRead == 1) && (yRead == 1))
	{
		densityBuffer[0] = 0.0f;
		densityBuffer[gGridSizeExternal - 1] = 0.0f;
		densityBuffer[gGridSizeExternal - 1 + strideDensity * (gGridSizeExternal - 1)] = 0.0f;
		densityBuffer[strideDensity * (gGridSizeExternal - 1)] = 0.0f;
	}

	if  (despawnData[xRead + despawnStride * yRead])
	{
		// Eliminate density on targets.
		densityBuffer[xRead + strideDensity * yRead] = 0.0f;
		
	} else
	{
		// Build the maximum, there may already be other people walking through the spawn area.
		float spawnValue = spawnArea[xRead + yRead * strideSpawn];
		if (spawnValue > 0.001f)
			densityBuffer[xRead + strideDensity * yRead] = fmaxf(densityBuffer[xRead + strideDensity * yRead], spawnValue);
	}

	

}

void DensityManager::EnforceBoundaryConditions(DataBase* dataBase)
{
	ApplyConditions  CUDA_DECORATOR_LOGIC (m_density.m_array, m_density.m_stride,
		dataBase->GetSpawnData().m_array, dataBase->GetSpawnData().m_stride,
		dataBase->GetDespawnData().m_array, dataBase->GetDespawnData().m_stride);
}



void DensityManager::GenerateDensityVisualization(uchar4* textureMemory)
{
	VisualizationHelper::VisualizeScalarField(m_density, gMaximumDensity,  textureMemory);
}

void DensityManager::UpdateDensityField(float timePassed, FloatArray timeField,  DataBase* dataBase)
{
	m_continuitySolver.IntegrateEquation(timePassed, m_density,  timeField, dataBase);
}

void DensityManager::ResetDensity(DataBase* dataBase)
{
	TransferHelper::CopyDataFromTo(dataBase->GetInitialDensityData(), m_density);
}
