#include "hip/hip_runtime.h"
#include "DensityManager.h"
#include "GlobalConstants.h"
#include "CudaHelper.h"
#include <cassert>
#include <>
#include <math.h>

DensityManager::DensityManager()
{
	m_targetAreaData = NULL;
	m_spawnAreaData = NULL;
	m_densityBuffer[0] = m_densityBuffer[1] = NULL;
	m_doubleBufferDensity = 0;
}

void DensityManager::InitializeManager(const char* spawnAreaFile, const char* targetAreaFile)
{
	assert(m_targetAreaData == NULL);

	m_targetAreaReader.ReadFile(targetAreaFile);
	m_spawnAreaReader.ReadFile(spawnAreaFile);

	m_spawnAreaData = m_transferHelper.UploadPictureAsFloat(&m_spawnAreaReader, 0.0f, 0.0f, gMaximumDensity, m_spawnAreaStride);
	m_targetAreaData = m_transferHelper.UploadPicture(&m_targetAreaReader, 0, m_targetAreaStride);
	m_densityBuffer[0] = m_transferHelper.ReserveFloatMemory(m_densityStride);
	m_densityBuffer[1] = m_transferHelper.ReserveFloatMemory(m_densityStride);
}

__global__ void ApplyConditions(float* densityBuffer, size_t strideDensity, unsigned* wallInformaton, size_t strideWall, float* spawnArea, size_t strideSpawn,
                    unsigned int* targetAreaData, size_t strideTarget)
{
	int xRead = threadIdx.x + blockIdx.x * blockDim.x + 1;
	int yRead = threadIdx.y + blockIdx.y * blockDim.y + 1;


	// Deal with the border cases.
	if (xRead == 1)
		densityBuffer[yRead * strideDensity] = 0.0f;
	if (yRead == 1)
		densityBuffer[xRead] = 0.0f;
	if (xRead == gGridSizeExternal - 2)
		densityBuffer[(gGridSizeExternal - 1) + yRead * strideDensity] = 0.0f;
	if (yRead == gGridSizeExternal - 2)
		densityBuffer[xRead + (gGridSizeExternal - 1) * strideDensity] = 0.0f;


	// Special fix for the corner cases.
	if ((xRead == 1) && (yRead == 1))
	{
		densityBuffer[0] = 0.0f;
		densityBuffer[gGridSizeExternal - 1] = 0.0f;
		densityBuffer[gGridSizeExternal - 1 + strideDensity * (gGridSizeExternal - 1)] = 0.0f;
		densityBuffer[strideDensity * (gGridSizeExternal - 1)] = 0.0f;
	}

	if ((wallInformaton[xRead + strideWall * yRead]) || (targetAreaData[xRead + strideTarget * yRead]))
	{
		// Wliminate density on walls and targets.
		densityBuffer[xRead + strideDensity * yRead] = 0.0f;
	} else
	{
		// Build the maximum, there may already be other people walking through the spawn area.
		float spawnValue = spawnArea[xRead + yRead * strideSpawn];
		if (spawnValue > 0.001f)
			densityBuffer[xRead + strideDensity * yRead] = fmaxf(densityBuffer[xRead + strideDensity * yRead], spawnValue);
	}

	

}

void DensityManager::EnforceBoundaryConditions(unsigned* wallInformation, size_t wallStride)
{
	ApplyConditions  CUDA_DECORATOR_LOGIC (m_densityBuffer[m_doubleBufferDensity], m_densityStride, wallInformation, wallStride,
		m_spawnAreaData, m_spawnAreaStride, m_targetAreaData, m_targetAreaStride);
}

void DensityManager::GenerateDensityVisualization(uchar4* textureMemory)
{
	m_transferHelper.VisualizeScalarField(m_densityBuffer[m_doubleBufferDensity], gMaximumDensity, m_densityStride, textureMemory);
}
