#include "hip/hip_runtime.h"
#include "TransferHelper.h"
#include "TgaReader.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include "CudaHelper.h"
#include <cstring>


int   TransferHelper::m_intArea[gGridSizeExternal * gGridSizeExternal];
float TransferHelper::m_floatArea[gGridSizeExternal * gGridSizeExternal];

UnsignedArray TransferHelper::UploadPicture(TgaReader* reader, unsigned char boundaryValue)
{
	assert(reader->GetWidth() == gGridSizeInternal);
	assert(reader->GetHeight() == gGridSizeInternal);

	
	
	unsigned char* internalPixelInformation = reader->GetPixels();

	for(int row = 0; row < gGridSizeExternal; ++row)
		for(int column = 0; column < gGridSizeExternal; ++column)
		{	
			unsigned char destinationValue = boundaryValue;
			if ((row != 0) && (row != gGridSizeExternal - 1) && (column != 0) && (column != gGridSizeExternal - 1))
				destinationValue = internalPixelInformation[3 * ((column - 1) + gGridSizeInternal * (row - 1))];
			m_intArea[column + row * gGridSizeExternal] = destinationValue;
		}

	// Allocate device memory.
	void* memory;
	size_t pitch;
	
	hipMallocPitch(&memory, &pitch, gGridSizeExternal * 4, gGridSizeExternal);
	hipMemcpy2D(memory, pitch, m_intArea, 4 * gGridSizeExternal, 4 * gGridSizeExternal, gGridSizeExternal, hipMemcpyHostToDevice);

	pitch /= 4;

	UnsignedArray result;
	result.m_array = (unsigned int*)memory;
	result.m_stride = pitch;

	return result;
}


FloatArray TransferHelper::UploadPictureAsFloat(TgaReader* reader, float boundaryValue, float minValueMapped,
	float maxValueMapped)
{
	assert(reader->GetWidth() == gGridSizeInternal);
	assert(reader->GetHeight() == gGridSizeInternal);

	unsigned char* internalPixelInformation = reader->GetPixels();

	for (int row = 0; row < gGridSizeExternal; ++row)
		for (int column = 0; column < gGridSizeExternal; ++column)
		{
			float destinationValue = boundaryValue;
			if ((row != 0) && (row != gGridSizeExternal - 1) && (column != 0) && (column != gGridSizeExternal - 1))
			{
				destinationValue = internalPixelInformation[3 * ((column - 1) + gGridSizeInternal * (row - 1))];
				destinationValue = minValueMapped + (maxValueMapped - minValueMapped) * destinationValue / 255.0f;
			}
			m_floatArea[column + row * gGridSizeExternal] = destinationValue;
		}

	// Allocate device memory.
	void* memory;
	size_t pitch;

	hipMallocPitch(&memory, &pitch, gGridSizeExternal * 4, gGridSizeExternal);
	hipMemcpy2D(memory, pitch, m_floatArea, 4 * gGridSizeExternal, 4 * gGridSizeExternal, gGridSizeExternal, hipMemcpyHostToDevice);

	pitch /= 4;
	FloatArray result;
	result.m_array = (float*)memory;
	result.m_stride = pitch;

	return result;
}

FloatArray TransferHelper::ReserveFloatMemory()
{
	// Allocate device memory.
	void* memory;
	size_t pitch;
	hipMallocPitch(&memory, &pitch, gGridSizeExternal * 4, gGridSizeExternal);

	// We fill all with zero at the beginnig.
	memset(m_floatArea, 0, gGridSizeExternal * gGridSizeExternal * 4);
	hipMemcpy2D(memory, pitch, m_floatArea, 4 * gGridSizeExternal, 4 * gGridSizeExternal, gGridSizeExternal, hipMemcpyHostToDevice);
	
	pitch /= 4;

	FloatArray result;
	result.m_array = (float*)memory;
	result.m_stride = pitch;
	
	return result;
}

UnsignedArray TransferHelper::ReserveUnsignedMemory()
{
	// Allocate device memory.
	void* memory;
	size_t pitch;
	hipMallocPitch(&memory, &pitch, gGridSizeExternal * 4, gGridSizeExternal);

	// We fill all with zero at the beginnig.
	memset(m_intArea, 0, gGridSizeExternal * gGridSizeExternal * 4);
	hipMemcpy2D(memory, pitch, m_intArea, 4 * gGridSizeExternal, 4 * gGridSizeExternal, gGridSizeExternal, hipMemcpyHostToDevice);

	pitch /= 4;

	UnsignedArray result;
	result.m_array = (unsigned int*)memory;
	result.m_stride = pitch;

	return result;
}

FloatArray TransferHelper::UpfronFilledValue(float value)
{
	for (int row = 0; row < gGridSizeExternal; ++row)
		for (int column = 0; column < gGridSizeExternal; ++column)
		{
			m_floatArea[column + row * gGridSizeExternal] = value;
		}

	// Allocate device memory.
	void* memory;
	size_t pitch;

	hipMallocPitch(&memory, &pitch, gGridSizeExternal * 4, gGridSizeExternal);
	hipMemcpy2D(memory, pitch, m_floatArea, 4 * gGridSizeExternal, 4 * gGridSizeExternal, gGridSizeExternal, hipMemcpyHostToDevice);

	pitch /= 4;
	FloatArray result;
	result.m_array = (float*)memory;
	result.m_stride = pitch;

	return result;
}

FloatArray TransferHelper::BuildHorizontalGradient(float startMax, int direction)
{
	for (int row = 0; row < gGridSizeExternal; ++row)
		for (int column = 0; column < gGridSizeExternal; ++column)
		{
			float destinationValue;
			if (direction == 1)
				destinationValue = startMax * ((float)column) / (gGridSizeExternal - 1);
			else
				destinationValue = startMax - startMax * ((float)column) / (gGridSizeExternal - 1);
			m_floatArea[column + row * gGridSizeExternal] = destinationValue;
		}

	// Allocate device memory.
	void* memory;
	size_t pitch;

	hipMallocPitch(&memory, &pitch, gGridSizeExternal * 4, gGridSizeExternal);
	hipMemcpy2D(memory, pitch, m_floatArea, 4 * gGridSizeExternal, 4 * gGridSizeExternal, gGridSizeExternal, hipMemcpyHostToDevice);

	pitch /= 4;
	FloatArray result;
	result.m_array = (float*)memory;
	result.m_stride = pitch;

	return result;
}

FloatArray TransferHelper::BuildVerticalGradient(float startMax, int direction)
{
	for (int row = 0; row < gGridSizeExternal; ++row)
		for (int column = 0; column < gGridSizeExternal; ++column)
		{
			float destinationValue;
			if (direction == 1)
				destinationValue = startMax * ((float)row) / (gGridSizeExternal - 1);
			else
				destinationValue = startMax - startMax * ((float)row) / (gGridSizeExternal - 1);
			m_floatArea[column + row * gGridSizeExternal] = destinationValue;
		}

	// Allocate device memory.
	void* memory;
	size_t pitch;

	hipMallocPitch(&memory, &pitch, gGridSizeExternal * 4, gGridSizeExternal);
	hipMemcpy2D(memory, pitch, m_floatArea, 4 * gGridSizeExternal, 4 * gGridSizeExternal, gGridSizeExternal, hipMemcpyHostToDevice);

	pitch /= 4;
	FloatArray result;
	result.m_array = (float*)memory;
	result.m_stride = pitch;

	return result;
}

FloatArray TransferHelper::BuildRadialGradient(float startMax, int direction)
{

	float maxDistance = sqrtf(2) * gGridSizeExternal / 2.0f;
	for (int row = 0; row < gGridSizeExternal; ++row)
		for (int column = 0; column < gGridSizeExternal; ++column)
		{
			float distance = sqrtf((float)((row - gGridSizeExternal / 2) * (row - gGridSizeExternal / 2) + (column - gGridSizeExternal / 2) * (column - gGridSizeExternal / 2)));
			distance /= maxDistance;
			if (direction == 1)
				distance = 1.0f - distance;

			m_floatArea[column + row * gGridSizeExternal] = startMax * distance;
		}

	// Allocate device memory.
	void* memory;
	size_t pitch;

	hipMallocPitch(&memory, &pitch, gGridSizeExternal * 4, gGridSizeExternal);
	hipMemcpy2D(memory, pitch, m_floatArea, 4 * gGridSizeExternal, 4 * gGridSizeExternal, gGridSizeExternal, hipMemcpyHostToDevice);

	pitch /= 4;
	FloatArray result;
	result.m_array = (float*)memory;
	result.m_stride = pitch;

	return result;
}











__global__ void CopyData(float* sourceArray, size_t sourceStride, float* destinationArray, size_t destinationStride)
{
	int baseX = (threadIdx.x + blockIdx.x * blockDim.x) + 1;
	int baseY = (threadIdx.y + blockIdx.y * blockDim.y) + 1;

	destinationArray[baseX + baseY * destinationStride] = sourceArray[baseX + baseY * sourceStride];

	// Here we have to deal with the boundaries.
	if (baseX == 1)
	{
		destinationArray[baseY * destinationStride] = sourceArray[ baseY * sourceStride];
	}
	if (baseY == 1)
	{
		destinationArray[baseX ] = sourceArray[baseX ];
	}
	if (baseX == gGridSizeExternal - 2)
	{
		destinationArray[(gGridSizeExternal - 1) + baseY * destinationStride] = sourceArray[(gGridSizeExternal - 1) + baseY * sourceStride];
	}
	if (baseY == gGridSizeExternal - 2)
	{
		destinationArray[baseX + (gGridSizeExternal - 1) * destinationStride] = sourceArray[baseX + (gGridSizeExternal - 1) * sourceStride];
	}

	// The 4 corner cases.
	if ((baseX == 1) && (baseY == 1))
	{
		destinationArray[0] = sourceArray[0];
		destinationArray[gGridSizeExternal - 1] = sourceArray[gGridSizeExternal - 1];
		destinationArray[(gGridSizeExternal - 1) * destinationStride] = sourceArray[(gGridSizeExternal - 1) * sourceStride];
		destinationArray[(gGridSizeExternal - 1) + (gGridSizeExternal - 1) * destinationStride] = sourceArray[(gGridSizeExternal - 1) + (gGridSizeExternal - 1) * sourceStride];
	}
}

void TransferHelper::CopyDataFromTo(FloatArray source, FloatArray destination)
{
	assert(source.m_array);
	assert(destination.m_array);
	CopyData CUDA_DECORATOR_LOGIC (source.m_array, source.m_stride, destination.m_array, destination.m_stride);
}
