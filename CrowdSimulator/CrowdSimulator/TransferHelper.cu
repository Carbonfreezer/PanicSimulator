#include "hip/hip_runtime.h"
#include "TransferHelper.h"
#include "TgaReader.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include "CudaHelper.h"
#include <cstring>


UnsignedArray TransferHelper::UploadPicture(TgaReader* reader, unsigned char boundaryValue)
{
	assert(reader->GetWidth() == gGridSizeInternal);
	assert(reader->GetHeight() == gGridSizeInternal);

	
	
	unsigned char* internalPixelInformation = reader->GetPixels();

	for(int row = 0; row < gGridSizeExternal; ++row)
		for(int column = 0; column < gGridSizeExternal; ++column)
		{	
			unsigned char destinationValue = boundaryValue;
			if ((row != 0) && (row != gGridSizeExternal - 1) && (column != 0) && (column != gGridSizeExternal - 1))
				destinationValue = internalPixelInformation[3 * ((column - 1) + gGridSizeInternal * (row - 1))];
			m_intArea[column + row * gGridSizeExternal] = destinationValue;
		}

	// Allocate device memory.
	void* memory;
	size_t pitch;
	
	hipMallocPitch(&memory, &pitch, gGridSizeExternal * 4, gGridSizeExternal);
	hipMemcpy2D(memory, pitch, m_intArea, 4 * gGridSizeExternal, 4 * gGridSizeExternal, gGridSizeExternal, hipMemcpyHostToDevice);

	pitch /= 4;

	UnsignedArray result;
	result.m_array = (unsigned int*)memory;
	result.m_stride = pitch;

	return result;
}


FloatArray TransferHelper::UploadPictureAsFloat(TgaReader* reader, float boundaryValue, float minValueMapped,
	float maxValueMapped)
{
	assert(reader->GetWidth() == gGridSizeInternal);
	assert(reader->GetHeight() == gGridSizeInternal);

	unsigned char* internalPixelInformation = reader->GetPixels();

	for (int row = 0; row < gGridSizeExternal; ++row)
		for (int column = 0; column < gGridSizeExternal; ++column)
		{
			float destinationValue = boundaryValue;
			if ((row != 0) && (row != gGridSizeExternal - 1) && (column != 0) && (column != gGridSizeExternal - 1))
			{
				destinationValue = internalPixelInformation[3 * ((column - 1) + gGridSizeInternal * (row - 1))];
				destinationValue = minValueMapped + (maxValueMapped - minValueMapped) * destinationValue / 255;
			}
			m_floatArea[column + row * gGridSizeExternal] = destinationValue;
		}

	// Allocate device memory.
	void* memory;
	size_t pitch;

	hipMallocPitch(&memory, &pitch, gGridSizeExternal * 4, gGridSizeExternal);
	hipMemcpy2D(memory, pitch, m_floatArea, 4 * gGridSizeExternal, 4 * gGridSizeExternal, gGridSizeExternal, hipMemcpyHostToDevice);

	pitch /= 4;
	FloatArray result;
	result.m_array = (float*)memory;
	result.m_stride = pitch;

	return result;
}

FloatArray TransferHelper::ReserveFloatMemory()
{
	// Allocate device memory.
	void* memory;
	size_t pitch;
	hipMallocPitch(&memory, &pitch, gGridSizeExternal * 4, gGridSizeExternal);

	// We fill all with zero at the beginnig.
	memset(m_floatArea, 0, gGridSizeExternal * gGridSizeExternal * 4);
	hipMemcpy2D(memory, pitch, m_floatArea, 4 * gGridSizeExternal, 4 * gGridSizeExternal, gGridSizeExternal, hipMemcpyHostToDevice);
	
	pitch /= 4;

	FloatArray result;
	result.m_array = (float*)memory;
	result.m_stride = pitch;
	
	return result;
}


__global__  void MarcIfFlagged(unsigned int* deviceMemory, size_t devicePitch, uchar4* pixelMemory, uchar4 color)
{
	int baseX = (threadIdx.x + blockIdx.x * blockDim.x) * gPixelsPerCell;
	int baseY = (threadIdx.y + blockIdx.y * blockDim.y) * gPixelsPerCell;
	for(int i = 0; i < gPixelsPerCell; ++i)
		for(int j = 0; j < gPixelsPerCell; ++j)
		{
			int srcX = i + baseX;
			int srcY = j + baseY;

			
			unsigned int candidate = deviceMemory[(srcX / gPixelsPerCell + 1) + devicePitch * (srcY / gPixelsPerCell + 1)];
			if (candidate)
				pixelMemory[srcX + gScreenResolution * srcY] = color;
		

		}
}

void TransferHelper::MarcColor(UnsignedArray data, uchar4* pixelMemory, uchar4 color)
{
	MarcIfFlagged CUDA_DECORATOR_LOGIC (data.m_array, data.m_stride, pixelMemory, color);
}


__global__ void  VisualizeField(float* deviceMemory, size_t devicePitch, float maximumValue, uchar4* pixelMemory)
{
	int baseX = (threadIdx.x + blockIdx.x * blockDim.x) ;
	int baseY = (threadIdx.y + blockIdx.y * blockDim.y) ;


	float candidate = deviceMemory[(baseX + 1) + devicePitch * (baseY + 1)];

	unsigned char redColor  = (unsigned char) (255.0f * fminf(maximumValue, candidate) / maximumValue);
	uchar4 finalColor = make_uchar4(redColor, 0, 255 - redColor, 255);

	baseX *= gPixelsPerCell;
	baseY *= gPixelsPerCell;

	for (int i = 0; i < gPixelsPerCell; ++i)
		for (int j = 0; j < gPixelsPerCell; ++j)
			pixelMemory[i + baseX + gScreenResolution * (j + baseY)] = finalColor;
}

void TransferHelper::VisualizeScalarField(FloatArray deviceData, float maximumValue, 
                                          uchar4* pixelMemory)
{
	
	VisualizeField CUDA_DECORATOR_LOGIC (deviceData.m_array, deviceData.m_stride, maximumValue, pixelMemory);
}


__global__ void  VisualizeFieldWithNegative(float* deviceMemory, size_t devicePitch, float maximumValue, uchar4* pixelMemory)
{
	int baseX = (threadIdx.x + blockIdx.x * blockDim.x);
	int baseY = (threadIdx.y + blockIdx.y * blockDim.y);


	float candidate = deviceMemory[(baseX + 1) + devicePitch * (baseY + 1)];
	candidate = fminf(candidate, maximumValue);
	candidate = fmaxf(candidate, -maximumValue);
	candidate = (candidate + maximumValue) / (2.0f * maximumValue);

	unsigned char redColor = (unsigned char)(255.0f * candidate);
	uchar4 finalColor = make_uchar4(redColor, 0, 255 - redColor, 255);

	baseX *= gPixelsPerCell;
	baseY *= gPixelsPerCell;

	for (int i = 0; i < gPixelsPerCell; ++i)
		for (int j = 0; j < gPixelsPerCell; ++j)
			pixelMemory[i + baseX + gScreenResolution * (j + baseY)] = finalColor;
}

void TransferHelper::VisualizeScalarFieldWithNegative(FloatArray deviceData, float maximumValue, 
	uchar4* pixelMemory)
{
	VisualizeFieldWithNegative CUDA_DECORATOR_LOGIC(deviceData.m_array, deviceData.m_stride, maximumValue, pixelMemory);
}


__global__ void GenerateLineFlags(float* dataMemory, size_t dataStride, unsigned int* isoLineFlags,
                                  size_t isoLineStride, float isoLineStepSize)
{
	__shared__ float valueBuffer[gBlockSize + 2][gBlockSize + 2];

	
	// We keep tack of the pixel block we are responsible for.
	int xOrigin = threadIdx.x + gBlockSize * blockIdx.x + 1;   
	int yOrigin = threadIdx.y + gBlockSize * blockIdx.y + 1;

	int xScan = threadIdx.x + 1;
	int yScan = threadIdx.y + 1;


	valueBuffer[xScan][yScan] = dataMemory[xOrigin  + yOrigin  * dataStride ];
	
	if (threadIdx.x == 0)
		valueBuffer[xScan - 1][yScan] = dataMemory[(xOrigin  - 1) + yOrigin  * dataStride ];
	if (threadIdx.x == 31)
		valueBuffer[xScan + 1][yScan] = dataMemory[(xOrigin + 1) + yOrigin  * dataStride ];
	if (threadIdx.y == 0)
		valueBuffer[xScan][yScan - 1 ] = dataMemory[xOrigin  + (yOrigin - 1) * dataStride ];
	if (threadIdx.y == 31)
		valueBuffer[xScan][yScan + 1] = dataMemory[xOrigin  + (yOrigin + 1) * dataStride ];
	
	__syncthreads();


	// Get nearest iso value.
	float localValue = valueBuffer[xScan][yScan];
	float nearestIsoValue = roundf(localValue / isoLineStepSize) * isoLineStepSize;

	bool linePlotting = false;

	linePlotting |= (valueBuffer[xScan - 1][yScan] > nearestIsoValue);
	linePlotting |= (valueBuffer[xScan + 1][yScan] > nearestIsoValue);
	linePlotting |= (valueBuffer[xScan][yScan + 1] > nearestIsoValue);
	linePlotting |= (valueBuffer[xScan][yScan - 1] > nearestIsoValue);

	linePlotting &= (localValue <= nearestIsoValue);

	isoLineFlags[xOrigin  + yOrigin  * isoLineStride] = linePlotting;
	
	
}

void TransferHelper::VisualizeIsoLines(FloatArray deviceData, float isoLineStepSize, 
                                       uchar4* pixelMemory)
{
	if (m_isoLineData.m_array == NULL)
		m_isoLineData = ReserveFloatMemory();

	dim3 block(32, 32);
	dim3 grid(3, 3);
	
	GenerateLineFlags CUDA_DECORATOR_LOGIC(deviceData.m_array, deviceData.m_stride, (unsigned int *)(m_isoLineData.m_array), m_isoLineData.m_stride,
	                                       isoLineStepSize);
	MarcIfFlagged CUDA_DECORATOR_LOGIC((unsigned int *)(m_isoLineData.m_array), m_isoLineData.m_stride, pixelMemory,
	                                   make_uchar4(128, 128, 128, 255));
}
